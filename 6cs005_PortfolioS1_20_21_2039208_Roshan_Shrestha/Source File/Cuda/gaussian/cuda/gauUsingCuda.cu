#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "lodepng.h"

#define width 500
#define height 500

//compile with c++ lodepng file

//nvcc GaussianFiltering.cu lodepng.cpp

unsigned int iwidth;
unsigned int iheight;

__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4);
  return image[i];
}

__device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +1;
  return image[i];
}

__device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +2;
  return image[i];
}

__device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +3;
  return image[i];
}

__device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red){
  unsigned int i = (row * width * 4) + (col * 4);
  image[i] = red;
}

__device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green){
  unsigned int i = (row * width * 4) + (col * 4) +1;
  image[i] = green;
}

__device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue){
  unsigned int i = (row * width * 4) + (col * 4) +2;
  image[i] = blue;
}

__device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha){
  unsigned int i = (row * width * 4) + (col * 4) +3;
  image[i] = alpha;
}

int time_difference(struct timespec *start, struct timespec *finish,
	long long int *difference)
{
	long long int ds = finish->tv_sec - start->tv_sec;
	long long int dn = finish->tv_nsec - start->tv_nsec;

	if (dn < 0)
	{
		ds--;
		dn += 1000000000;
	}
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}

//GPU function 
__global__ void gaussianFunction(unsigned char * gpu_imageOuput, unsigned char * gpu_imageInput){

	unsigned redTL,redTC, redTR;
	unsigned redL, redC, redR;
	unsigned redBL,redBC, redBR;
	unsigned newRed;

        unsigned greenTL,greenTC, greenTR;
        unsigned greenL, greenC, greenR;
        unsigned greenBL,greenBC, greenBR;
        unsigned newGreen;

	unsigned blueTL,blueTC, blueTR;
	unsigned blueL, blueC, blueR;
	unsigned blueBL,blueBC, blueBR;
	unsigned newBlue;
	  
	float filter[3][3] = {
	  { 1.0/16, 2.0/16, 1.0/16 },
	  { 2.0/16, 4.0/16, 2.0/16 },
	  { 1.0/16, 2.0/16, 1.0/16 }};

	int row = blockIdx.x+1;
	int col = threadIdx.x+1;
	
	
	setGreen(gpu_imageOuput, row, col, getGreen(gpu_imageInput, row, col));
        setBlue(gpu_imageOuput, row, col, getBlue(gpu_imageInput, row, col));
        setAlpha(gpu_imageOuput, row, col, 255);

        redTL = getRed(gpu_imageInput, row-1, col-1);
        redTC = getRed(gpu_imageInput, row-1, col);
        redTR = getRed(gpu_imageInput, row-1, col+1);

        redL = getRed(gpu_imageInput, row, col-1);
        redC = getRed(gpu_imageInput, row, col);
        redR = getRed(gpu_imageInput, row, col+1);

        redBL = getRed(gpu_imageInput, row+1, col-1);
        redBC = getRed(gpu_imageInput, row+1, col);
        redBR = getRed(gpu_imageInput, row+1, col+1);

        newRed = redTL*filter[0][0] + redTC*filter[0][1] + redTR*filter[0][2]
	     + redL*filter[1][0]  + redC*filter[1][1]  + redR*filter[1][2]
	     + redBL*filter[2][0] + redBC*filter[2][1] + redBR*filter[2][2];
 
        setRed(gpu_imageOuput, row, col, newRed);

        greenTL = getGreen(gpu_imageInput, row-1, col-1);
        greenTC = getGreen(gpu_imageInput, row-1, col);
        greenTR = getGreen(gpu_imageInput, row-1, col+1);
  
        greenL = getGreen(gpu_imageInput, row, col-1);
        greenC = getGreen(gpu_imageInput, row, col);
        greenR = getGreen(gpu_imageInput, row, col+1);

        greenBL = getGreen(gpu_imageInput, row+1, col-1);
        greenBC = getGreen(gpu_imageInput, row+1, col);
        greenBR = getGreen(gpu_imageInput, row+1, col+1);

        newGreen = greenTL*filter[0][0] + greenTC*filter[0][1] + greenTR*filter[0][2]
	     + greenL*filter[1][0]  + greenC*filter[1][1]  + greenR*filter[1][2]
	     + greenBL*filter[2][0] + greenBC*filter[2][1] + greenBR*filter[2][2];
 
        setGreen(gpu_imageOuput, row, col, newGreen);

        blueTL = getBlue(gpu_imageInput, row-1, col-1);
        blueTC = getBlue(gpu_imageInput, row-1, col);
        blueTR = getBlue(gpu_imageInput, row-1, col+1);

        blueL = getBlue(gpu_imageInput, row, col-1);
        blueC = getBlue(gpu_imageInput, row, col);
        blueR = getBlue(gpu_imageInput, row, col+1);

        blueBL = getBlue(gpu_imageInput, row+1, col-1);
        blueBC = getBlue(gpu_imageInput, row+1, col);
        blueBR = getBlue(gpu_imageInput, row+1, col+1);

        newBlue = blueTL*filter[0][0] + blueTC*filter[0][1] + blueTR*filter[0][2]
	     + blueL*filter[1][0]  + blueC*filter[1][1]  + blueR*filter[1][2]
	     + blueBL*filter[2][0] + blueBC*filter[2][1] + blueBR*filter[2][2];
 
        setBlue(gpu_imageOuput, row, col, newBlue);
        
}

int main(int argc, char **argv){

	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	const char* filename = argv[1];
	const char* newFileName = "gaussian_filtered.png";
	
	
	error = lodepng_decode32_file(&image, &iwidth, &iheight, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}
	
	
	printf("Image width = %d height = %d\n", iwidth, iheight);

	const int ARRAY_SIZE = iwidth*iheight*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	//Start Timer
	struct timespec start, finish;
	long long int time_elapsed;

	clock_gettime(CLOCK_MONOTONIC, &start);
	
	// launch the kernel
	gaussianFunction<<< iheight-2, iwidth-2 >>>(d_out, d_in);

	hipDeviceSynchronize();	
	
	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	printf("%s\n",host_imageOutput);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, iwidth, iheight);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}
	
	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lld ns or %0.2lf sec\n", time_elapsed, (time_elapsed / 1.0e9));

	//free(image);
	//free(host_imageInput);
	//hipFree(d_in);
	//hipFree(d_out);

	return 0;
}
